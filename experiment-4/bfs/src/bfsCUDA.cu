
#include <hip/hip_runtime.h>


extern "C" {
// __global__ void bfsKernel(const int* adjacencyList, const int* edgesOffset,
// const int* edgesSize,
//                             const int* currentQueue, int* nextQueue, int*
//                             distance, int* parent, int* nextQueueSize, int
//                             currentQueueSize)
// {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;    // 每个线程处理
//     currentQueue 中的一个节点 if (tid >= currentQueueSize) return; int node =
//     currentQueue[tid];           // 当前线程处理的节点 int nodeDistance =
//     distance[node]; int start = edgesOffset[node];          //
//     遍历该节点的邻居 int end = start + edgesSize[node]; for (int i = start; i
//     < end; ++i)
//     {
//         int neighbor = adjacencyList[i];
//         if (atomicCAS(&distance[neighbor], INT_MAX, nodeDistance + 1) ==
//         INT_MAX)
//         {
//             parent[neighbor] = node;        // 设置邻居的父节点
//             int index = atomicAdd(nextQueueSize, 1);
//             nextQueue[index] = neighbor;
//         }
//     }
// }
__global__ void bfsKernel(const int *adjacencyList, const int *edgesOffset,
                          const int *edgesSize, const int *currentQueue,
                          int *nextQueue, int *distance, int *parent,
                          int *nextQueueSize, int currentQueueSize) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int warpID = bid * 4 + (tid >> 5);
  int laneID = tid & 31;
  if (warpID >= currentQueueSize)
    return;
  int node = currentQueue[warpID]; // 当前线程处理的节点
  int nodeDistance = distance[node];
  int start = edgesOffset[node]; // 遍历该节点的邻居
  int end = start + edgesSize[node];
  for (int i = start + laneID; i < end; i += warpSize) {
    int neighbor = adjacencyList[i];
    if (atomicCAS(&distance[neighbor], INT_MAX, nodeDistance + 1) == INT_MAX) {
      parent[neighbor] = node; // 设置邻居的父节点
      int index = atomicAdd(nextQueueSize, 1);
      nextQueue[index] = neighbor;
    }
  }
}
}

#include "hip/hip_runtime.h"
#include <cstdio>
#include <string>

#include <hip/hip_runtime.h>

#include "bfsCPU.h"
#include "bfsCUDA.cu"
#include "graph.h"
#include "hwtimer.h"

#define GPU_DEVICE 0

void runCpu(int startVertex, Graph &G, std::vector<int> &distance,
            std::vector<int> &parent, std::vector<bool> &visited) {
  bfsCPU(startVertex, G, distance, parent, visited);
}

void checkError(hipError_t error, std::string msg) {
  if (error != hipSuccess) {
    printf("%s: %d\n", msg.c_str(), error);
    exit(1);
  }
}

hipDeviceProp_t deviceProp;

int *d_adjacencyList;
int *d_edgesOffset;
int *d_edgesSize;
int *d_distance;
int *d_parent;
int *d_currentQueue;
int *d_nextQueue;
int *d_degrees;
int *incrDegrees;

void initCuda(Graph &G) {
  // initialize CUDA
  checkError(hipGetDeviceProperties(&deviceProp, GPU_DEVICE),
             "cannot get device");
  printf("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
  checkError(hipSetDevice(GPU_DEVICE), "cannot set device");

  // copy memory to device
  checkError(hipMalloc(&d_adjacencyList, G.numEdges * sizeof(int)),
             "cannot allocate d_adjacencyList");
  checkError(hipMalloc(&d_edgesOffset, G.numVertices * sizeof(int)),
             "cannot allocate d_edgesOffset");
  checkError(hipMalloc(&d_edgesSize, G.numVertices * sizeof(int)),
             "cannot allocate d_edgesSize");
  checkError(hipMalloc(&d_distance, G.numVertices * sizeof(int)),
             "cannot allocate d_distance");
  checkError(hipMalloc(&d_parent, G.numVertices * sizeof(int)),
             "cannot allocate d_parent");
  checkError(hipMalloc(&d_currentQueue, G.numVertices * sizeof(int)),
             "cannot allocate d_currentQueue");
  checkError(hipMalloc(&d_nextQueue, G.numVertices * sizeof(int)),
             "cannot allocate d_nextQueue");
  checkError(hipMalloc(&d_degrees, G.numVertices * sizeof(int)),
             "cannot allocate d_degrees");
  checkError(hipHostMalloc((void **)&incrDegrees, sizeof(int) * G.numVertices),
             "cannot allocate memory");

  checkError(hipMemcpy(d_adjacencyList, G.adjacencyList.data(),
                        G.numEdges * sizeof(int), hipMemcpyHostToDevice),
             "cannot copy to d_adjacencyList");
  checkError(hipMemcpy(d_edgesOffset, G.edgesOffset.data(),
                        G.numVertices * sizeof(int), hipMemcpyHostToDevice),
             "cannot copy to d_edgesOffset");
  checkError(hipMemcpy(d_edgesSize, G.edgesSize.data(),
                        G.numVertices * sizeof(int), hipMemcpyHostToDevice),
             "cannot copy to d_edgesSize");
}

void finalizeCuda() {
  // free memory
  checkError(hipFree(d_adjacencyList),
             "cannot free memory for d_adjacencyList");
  checkError(hipFree(d_edgesOffset), "cannot free memory for d_edgesOffset");
  checkError(hipFree(d_edgesSize), "cannot free memory for d_edgesSize");
  checkError(hipFree(d_distance), "cannot free memory for d_distance");
  checkError(hipFree(d_parent), "cannot free memory for d_parent");
  checkError(hipFree(d_currentQueue), "cannot free memory for d_parent");
  checkError(hipFree(d_nextQueue), "cannot free memory for d_parent");
  checkError(hipHostFree(incrDegrees), "cannot free memory for incrDegrees");
}

void checkOutput(std::vector<int> &distance, std::vector<int> &expectedDistance,
                 Graph &G) {
  for (int i = 0; i < G.numVertices; i++) {
    if (distance[i] != expectedDistance[i]) {
      printf("%d %d %d\n", i, distance[i], expectedDistance[i]);
      printf("Wrong output!\n");
      exit(1);
    }
  }

  printf("Output OK!\n\n");
}

void initializeCudaBfs(int startVertex, std::vector<int> &distance,
                       std::vector<int> &parent, Graph &G) {
  // initialize values
  std::fill(distance.begin(), distance.end(), std::numeric_limits<int>::max());
  std::fill(parent.begin(), parent.end(), std::numeric_limits<int>::max());
  distance[startVertex] = 0;
  parent[startVertex] = 0;

  checkError(hipMemcpy(d_distance, distance.data(),
                        G.numVertices * sizeof(int), hipMemcpyHostToDevice),
             "cannot copy to d)distance");
  checkError(hipMemcpy(d_parent, parent.data(), G.numVertices * sizeof(int),
                        hipMemcpyHostToDevice),
             "cannot copy to d_parent");

  int firstElementQueue = startVertex;
  hipMemcpy(d_currentQueue, &firstElementQueue, sizeof(int),
             hipMemcpyHostToDevice);
}

void finalizeCudaBfs(std::vector<int> &distance, std::vector<int> &parent,
                     Graph &G) {
  // copy memory from device
  checkError(hipMemcpy(distance.data(), d_distance,
                        G.numVertices * sizeof(int), hipMemcpyDeviceToHost),
             "cannot copy d_distance to host");
  checkError(hipMemcpy(parent.data(), d_parent, G.numVertices * sizeof(int),
                        hipMemcpyDeviceToHost),
             "cannot copy d_parent to host");
}

void runCudaBfs(int startVertex, Graph &G, std::vector<int> &distance,
                std::vector<int> &parent) {
  initializeCudaBfs(startVertex, distance, parent, G);

  int blockSize = 128; // 每个线程块的线程数
  int *d_nextQueueSize;
  checkError(hipMalloc(&d_nextQueueSize, sizeof(int)),
             "cannot allocate d_nextQueueSize");

  int currentQueueSize = 1; // 初始队列大小为1（起始节点）
  while (currentQueueSize > 0) {
    // 初始化下一层队列大小为0
    checkError(hipMemset(d_nextQueueSize, 0, sizeof(int)),
               "cannot reset d_nextQueueSize");

    // 计算网格大小
    int gridSize = (currentQueueSize + blockSize / 32 - 1) / (blockSize / 32);

    // 启动 CUDA 核函数
    bfsKernel<<<gridSize, blockSize>>>(
        d_adjacencyList, d_edgesOffset, d_edgesSize, d_currentQueue,
        d_nextQueue, d_distance, d_parent, d_nextQueueSize, currentQueueSize);
    checkError(hipDeviceSynchronize(), "kernel launch failed");

    // 读取下一层队列大小
    checkError(hipMemcpy(&currentQueueSize, d_nextQueueSize, sizeof(int),
                          hipMemcpyDeviceToHost),
               "cannot copy nextQueueSize to host");

    // 交换队列指针
    std::swap(d_currentQueue, d_nextQueue);
  }

  hipFree(d_nextQueueSize); // 释放临时变量
  finalizeCudaBfs(distance, parent, G);
}

__global__ void multiply_by_two(int *d_data, int n) {
  int idx = threadIdx.x;
  if (idx < n)
    d_data[idx] += 2;
}
void cuda_warmup() {
  int VECTOR_SIZE = 100;
  int h_data[100];
  int *d_data;
  for (int i = 0; i < VECTOR_SIZE; i++)
    h_data[i] = i + 1;
  hipMalloc((void **)&d_data, VECTOR_SIZE * sizeof(int));
  hipMemcpy(d_data, h_data, VECTOR_SIZE * sizeof(int), hipMemcpyHostToDevice);
  for (int i = 0; i < 1000; i++)
    multiply_by_two<<<1, VECTOR_SIZE>>>(d_data, VECTOR_SIZE);
  hipMemcpy(h_data, d_data, VECTOR_SIZE * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_data);
}

int main(int argc, char **argv) {
  if (argc != 4) {
    printf("usage: ./bfs-exec <start vertex> <number of vertices> <number of "
           "edges>\n");
    exit(1);
  }

  // read graph from standard input
  Graph G;
  int startVertex = atoi(argv[1]);

  readGraph(G, argc, argv);

  printf("Number of vertices %d\n", G.numVertices);
  printf("Number of edges %d\n\n", G.numEdges);

  // vectors for results
  std::vector<int> distance(G.numVertices, std::numeric_limits<int>::max());
  std::vector<int> parent(G.numVertices, std::numeric_limits<int>::max());
  std::vector<bool> visited(G.numVertices, false);

  FILE *file = fopen("../data.txt", "a");

  HWTimer timer;

  printf("Starting sequential bfs.\n");

  const hwtime_t startTime = timer.get_time_ns();
  // run CPU sequential bfs
  runCpu(startVertex, G, distance, parent, visited);
  const hwtime_t endTime = timer.get_time_ns();

  const hwtime_t diff = endTime - startTime;
  const long diffInNS = diff.tv_nsec + ONE_S_TO_NS * diff.tv_sec;

  bool usingCUDA = false;
  fprintf(file, "%d %d %d %d %lu\n", usingCUDA, startVertex, G.numVertices,
          G.numEdges, diffInNS);

  printf("Elapsed time: %lu ns.\n\n", diffInNS);
  // save results from sequential bfs
  std::vector<int> expectedDistance(distance);
  std::vector<int> expectedParent(parent);

  // run CUDA simple parallel bfs
  printf("Starting parallel bfs.\n");

  initCuda(G);
  cuda_warmup();

  const hwtime_t startTimeCUDA = timer.get_time_ns();
  runCudaBfs(startVertex, G, distance, parent);
  const hwtime_t endTimeCUDA = timer.get_time_ns();
  const hwtime_t diffCUDA = endTimeCUDA - startTimeCUDA;
  const long diffCUDAInNS = diffCUDA.tv_nsec + ONE_S_TO_NS * diffCUDA.tv_sec;

  usingCUDA = true;
  fprintf(file, "%d %d %d %d %lu\n", usingCUDA, startVertex, G.numVertices,
          G.numEdges, diffCUDAInNS);

  printf("Elapsed time: %lu ns.\n\n", diffCUDAInNS);

  checkOutput(distance, expectedDistance, G);

  finalizeCuda();
  return 0;
}

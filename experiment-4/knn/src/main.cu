#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include <hip/hip_runtime_api.h>

#include "base.h"
#include "hwtimer.h"
#include "knnCPU.h"
#include "knnCUDA.cu"

void knnInit(float *coords, float *newCoords, int *classes, int numSamples,
             int numClasses, int numNewSamples);
void genRandCoords(float *x, int numSamples);
void checkOutput(float *classes, float *gpuClasses, int numClasses,
                 int totalSamples);

void checkOutput(int *classes, int *gpuClasses, int numClasses,
                 int totalSamples) {
  int *numCpuClasses = (int *)malloc(sizeof(int) * numClasses);
  int *numGpuClasses = (int *)malloc(sizeof(int) * numClasses);

  for (int j = 0; j < numClasses; j++) {
    numCpuClasses[j] = 0;
    numGpuClasses[j] = 0;
  }

  for (int i = 0; i < totalSamples; i++) {
    for (int j = 0; j < numClasses; j++) {
      if (classes[i] == j)
        numCpuClasses[j] += 1;
      if (gpuClasses[i] == j)
        numGpuClasses[j] += 1;
    }
  }

  for (int j = 0; j < numClasses; j++) {
    if (numCpuClasses[j] != numGpuClasses[j]) {
      printf("Wrong output!\n");
      exit(1);
    }
  }

  printf("Output OK!\n\n");
}

void knnInit(float *coords, float *newCoords, int *classes, int numSamples,
             int numClasses, int numNewSamples) {
  for (int i = 0; i < numSamples; i++)
    classes[i] = rand() % numClasses;

  genRandCoords(coords, numSamples);
  genRandCoords(newCoords, numNewSamples);
}
__global__ void multiply_by_two(int *d_data, int n) {
  int idx = threadIdx.x;
  if (idx < n)
    d_data[idx] += 2;
}
void cuda_warmup() {
  int VECTOR_SIZE = 100;
  int h_data[100];
  int *d_data;
  for (int i = 0; i < VECTOR_SIZE; i++)
    h_data[i] = i + 1;
  hipMalloc((void **)&d_data, VECTOR_SIZE * sizeof(int));
  hipMemcpy(d_data, h_data, VECTOR_SIZE * sizeof(int), hipMemcpyHostToDevice);
  for (int i = 0; i < 1000; i++)
    multiply_by_two<<<1, VECTOR_SIZE>>>(d_data, VECTOR_SIZE);
  hipMemcpy(h_data, d_data, VECTOR_SIZE * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_data);
}
void genRandCoords(float *x, int numSamples) {
  for (int i = 0; i < numSamples; i++)
    for (int j = 0; j < DIMENSION; j++)
      x[i * DIMENSION + j] =
          (float)rand() / (float)(RAND_MAX / POINTS_MAX) + (float)(POINTS_MIN);
}

int main(int argc, char **argv) {
  if (argc != 5) {
    printf("usage: ./knn_exec <k nearest neighbors> <number of classes> "
           "<number of existing samples> <number of new samples>\n");
    exit(1);
  }
  int device;
  hipDeviceProp_t prop;

  // 获取当前使用的设备号
  hipGetDevice(&device);

  // 获取设备的属性
  hipGetDeviceProperties(&prop, device);

  // 打印每个维度支持的最大 Grid 尺寸
  printf("Max grid size:\n");
  printf("  X dimension: %d\n", prop.maxGridSize[0]);
  printf("  Y dimension: %d\n", prop.maxGridSize[1]);
  printf("  Z dimension: %d\n", prop.maxGridSize[2]);
  printf("Max shared memory per block: %ld bytes\n", prop.sharedMemPerBlock);
  hwtimer_t timer;
  initTimer(&timer);

  int k = atoi(argv[1]);             // number of k nearest neighbors
  int numClasses = atoi(argv[2]);    // number of classes
  int numSamples = atoi(argv[3]);    // number of existing samples
  int numNewSamples = atoi(argv[4]); // number of samples to classify
  int numTotalSamples = numSamples + numNewSamples; // total samples

  // array with a class for each sample
  int *classes;
  float *newCoords;
  float *coords;
  hipHostMalloc((void **)&classes, sizeof(int) * numTotalSamples);
  hipHostMalloc((void **)&newCoords,
                 sizeof(float) * numNewSamples * DIMENSION);
  hipHostMalloc((void **)&coords, sizeof(float) * numTotalSamples * DIMENSION);

  // gpu samples (initialized from cpu samples)
  int *gpuClasses = (int *)malloc(sizeof(int) * numTotalSamples);
  float *gpuNewCoords =
      (float *)malloc(sizeof(float) * numNewSamples * DIMENSION);
  float *gpuCoords =
      (float *)malloc(sizeof(float) * numTotalSamples * DIMENSION);

  srand(12345);

  printf("Starting initialization.\n");
  startTimer(&timer);
  knnInit(coords, newCoords, classes, numSamples, numClasses, numNewSamples);
  stopTimer(&timer);
  printf("Elapsed time: %lu ns.\n\n", getTimerNs(&timer));

  memcpy(gpuClasses, classes, sizeof(int) * numTotalSamples);
  memcpy(gpuNewCoords, newCoords, sizeof(float) * numNewSamples * DIMENSION);
  memcpy(gpuCoords, coords, sizeof(float) * numTotalSamples * DIMENSION);
  FILE *file = fopen("../data.txt", "a");
  printf("Starting sequential knn.\n");
  startTimer(&timer);
  knnSerial(coords, newCoords, classes, numClasses, numSamples, numNewSamples,
            k);
  stopTimer(&timer);
  fprintf(file, "%d %d %d %d %lu ", k, numClasses, numSamples, numNewSamples,
          getTimerNs(&timer));

  printf("Elapsed time: %lu ns.\n\n", getTimerNs(&timer));
  cuda_warmup();
  printf("Starting parallel knn.\n");
  startTimer(&timer);
  knnParallel(gpuCoords, gpuNewCoords, gpuClasses, numClasses, numSamples,
              numNewSamples, k);
  stopTimer(&timer);
  printf("Elapsed time: %lu ns.\n\n", getTimerNs(&timer));
  fprintf(file, "%lu\n", getTimerNs(&timer));
  checkOutput(classes, gpuClasses, numClasses, numTotalSamples);
  fclose(file);
  hipHostFree(classes);
  hipHostFree(newCoords);
  hipHostFree(coords);
  free(gpuClasses);
  free(gpuNewCoords);
  free(gpuCoords);

  return 0;
}

#include "hip/hip_runtime.h"
#include <cfloat>
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime_api.h>

#include <thrust/binary_search.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/scan.h>
#include <thrust/sort.h>

#include "base.h"

void check_error(hipError_t err, const char *msg);
void knnParallel(float *coords, float *newCoords, int *classes, int numClasses,
                 int numSamples, int numNewSamples, int k);

__global__ void kernelknn(float *coords, float *newCoords, int *classes,
                          int numClasses, int numSamples, int numNewSamples,
                          int k) {
  int bid = blockIdx.x;  // 每个 block 负责一个新样本点
  int tid = threadIdx.x; // 每个线程负责一个旧样本点
  __shared__ float distances[10000];
  __shared__ float sharedneighdist[400];
  __shared__ int sharedneighclass[400];
  float neighdist[20];
  int neighclass[20];
  for (int i = 0; i < k; i++) {
    neighdist[i] = FLT_MAX;
    neighclass[i] = -1;
  }
  for (int t = 0; t < numSamples / 10000; t++) {
    for (int i = tid; i < 10000; i += blockDim.x) {
      float distance = 0.0;
      for (int d = 0; d < DIMENSION; d++) {
        float diff = newCoords[bid * DIMENSION + d] -
                     coords[(i + t * 10000) * DIMENSION + d];
        distance += diff * diff;
      }
      distances[i] = distance;
    }
    __syncthreads();
    if (tid < 20) {
      for (int i = 0; i < 500; i++) {
        if (distances[i + tid * 500] < neighdist[k - 1]) {
          neighdist[k - 1] = distances[i + tid * 500];
          neighclass[k - 1] = classes[i + tid * 500 + t * 10000];
          for (int m = k - 1; m > 0 && neighdist[m] < neighdist[m - 1]; m--) {
            float tempdist = neighdist[m];
            neighdist[m] = neighdist[m - 1];
            neighdist[m - 1] = tempdist;
            int tempclass = neighclass[m];
            neighclass[m] = neighclass[m - 1];
            neighclass[m - 1] = tempclass;
          }
        }
      }
    }
    __syncthreads();
  }
  if (tid < 20) {
    for (int i = 0; i < k; i++) {
      sharedneighdist[tid * k + i] = neighdist[i];
      sharedneighclass[tid * k + i] = neighclass[i];
    }
  }
  __shared__ float minElements[20];
  __shared__ int minClasses[20];

  // 初始化最小元素和类别数组
  if (tid < k) {
    minElements[tid] = FLT_MAX;
    minClasses[tid] = -1; // 假设类别从-1开始
  }

  // 找到前K个最小的元素
  if (tid == 0)
    for (int i = 0; i < 20 * k; i++) {
      float currentDist = sharedneighdist[i];
      int currentClass = sharedneighclass[i];

      // 插入新元素
      for (int j = 0; j < k; j++) {
        if (currentDist < minElements[j]) {
          // 插入新元素，并保持顺序
          for (int l = k - 1; l > j; l--) {
            minElements[l] = minElements[l - 1];
            minClasses[l] = minClasses[l - 1];
          }
          minElements[j] = currentDist;
          minClasses[j] = currentClass;
          break;
        }
      }
    }
  __shared__ int classCounts[20];
  if (tid < numClasses)
    classCounts[tid] = 0;
  __syncthreads();
  if (tid < k)
    atomicAdd(&classCounts[minClasses[tid]], 1);
  if (tid == 0) {
    int maxCount = 0;
    int predictedClass = 0;
    for (int c = 0; c < numClasses; c++) {
      if (classCounts[c] > maxCount) {
        maxCount = classCounts[c];
        predictedClass = c;
      }
    }
    classes[numSamples + bid] = predictedClass;
  }
}

void knnParallel(float *coords, float *newCoords, int *classes, int numClasses,
                 int numSamples, int numNewSamples, int k) {
  float *d_coords;
  float *d_newCoords;
  int *d_classes;

  int totalSamples = numSamples + numNewSamples;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  check_error(hipMalloc(&d_coords, numSamples * DIMENSION * sizeof(float)),
              "alloc d_coords");
  check_error(
      hipMalloc(&d_newCoords, numNewSamples * DIMENSION * sizeof(float)),
      "alloc d_newCoords");
  check_error(hipMalloc(&d_classes, totalSamples * sizeof(int)),
              "alloc d_classes");

  check_error(hipMemcpy(d_coords, coords,
                         numSamples * DIMENSION * sizeof(float),
                         hipMemcpyHostToDevice),
              "copy d_coords");
  check_error(hipMemcpy(d_newCoords, newCoords,
                         numNewSamples * DIMENSION * sizeof(float),
                         hipMemcpyHostToDevice),
              "copy d_newCoords");
  check_error(hipMemcpy(d_classes, classes, totalSamples * sizeof(int),
                         hipMemcpyHostToDevice),
              "copy d_classes");

  // 启动距离计算的 kernel
  int threadnum = 32 * 8;
  int blocknum = numNewSamples * (numSamples / numSamples);
  hipEventRecord(start); // 记录开始时间
  kernelknn<<<blocknum, threadnum>>>(d_coords, d_newCoords, d_classes,
                                     numClasses, numSamples, numNewSamples, k);
  hipDeviceSynchronize();
  hipEventRecord(stop); // 记录结束时间

  hipEventSynchronize(stop); // 确保事件完成

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop); // 计算时间差
  long long nanoseconds = (long long)(milliseconds * 1000000); // 转换为纳秒
  // printf("Elapsed time: %lld ns.\n", nanoseconds);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  // 复制结果回主机
  check_error(hipMemcpy(classes, d_classes, totalSamples * sizeof(int),
                         hipMemcpyDeviceToHost),
              "copy back classes");

  // 释放 GPU 内存
  hipFree(d_coords);
  hipFree(d_newCoords);
  hipFree(d_classes);
}

void check_error(hipError_t err, const char *msg) {
  if (err != hipSuccess) {
    fprintf(stderr, "%s : error %d (%s)\n", msg, err, hipGetErrorString(err));
    exit(err);
  }
}
